#include "hip/hip_runtime.h"
#include "sha256.cuh"

#include <chrono>
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <ostream>
#include <stdlib.h>
#include <string>
#include <unistd.h>

#define SHOW_INTERVAL_MS 2000
#define BLOCK_SIZE 256
#define SHA_PER_ITERATIONS 1'048'576
// #define SHA_PER_ITERATIONS 67'108'864
// #define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE
#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE

static size_t difficulty = 1;

static uint64_t nonce = 0;
static uint64_t user_nonce = 0;
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug infos
static std::chrono::high_resolution_clock::time_point t_last_updated;
__device__ bool checkZeroPadding(unsigned char *sha, uint8_t difficulty) {

  bool isOdd = difficulty % 2 != 0;
  uint8_t max = (difficulty / 2) + 1;

  /*
          Odd : 00 00 01 need to check 0 -> 2
          Even : 00 00 00 1 need to check 0 -> 3
          odd : 5 / 2 = 2 => 2 + 1 = 3
          even : 6 / 2 = 3 => 3 + 1 = 4
  */
  for (uint8_t cur_byte = 0; cur_byte < max; ++cur_byte) {
    uint8_t b = sha[cur_byte];
    if (cur_byte < max - 1) { // Before the last byte should be all zero
      if (b != 0)
        return false;
    } else if (isOdd) {
      if (b > 0x0F || b == 0)
        return false;
    } else if (b <= 0x0f)
      return false;
  }

  return true;
}
__device__ void generateRandomString(char *nonce) {
  hiprandState_t state;
  hiprand_init(clock64(), threadIdx.x, 0, &state); // 初始化随机数生成器

  int length = 10;
  const char charset[] = "abcdefghijklmnopqrstuvwxyz0"
                         "123456789"; // 字符集
  for (int i = 0; i < length; i++) {
    int index = hiprand_uniform(&state) * (sizeof(charset) - 1); // 生成随机索引
    nonce[i] = charset[index];
  }
  nonce[length] = '\0'; //
}

__global__ void sha256_kernel(char *out_input_string_nonce,
                              unsigned char *out_found_hash, int *out_found,
                              char *result_nonce, const char *in_input_string,
                              size_t in_input_string_size, uint8_t difficulty,
                              int offset) {

  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  char nonce[10 + 1];
  generateRandomString(nonce);
  // if (*out_found == 1) {
  //   return;
  // }
  unsigned char sha[32];
  {
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, in_input_string, in_input_string_size, nonce, offset);
    sha256_final(&ctx, sha);
  }

  if (checkZeroPadding(sha, difficulty) && atomicExch(out_found, 1) == 0) {
    memcpy(result_nonce, nonce, 10 + 1);
    memcpy(out_found_hash, sha, 32);
  }
}

void pre_sha256() {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0,
                                     hipMemcpyHostToDevice));
}

// Prints a 32 bytes sha256 to the hexadecimal form filled with zeroes
void print_hash(const unsigned char *sha256) {
  for (uint8_t i = 0; i < 32; ++i) {
    std::cout << std::hex << std::setfill('0') << std::setw(2)
              << static_cast<int>(sha256[i]);
  }
  std::cout << std::dec << std::endl;
}

const char *solve(std::string line, int difficulty) {
  // Init
  hipSetDevice(0);
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);
  // Get Offset
  std::string nonceString = "nonce";
  size_t found = line.find(nonceString);
  if (found == std::string::npos) {
    std::cout << "Substring found at index: " << found << std::endl;
    throw std::runtime_error("No nonce found line is" + line);
  }
  size_t offset = found + nonceString.size() + 3;
  //
  auto start_time = std::chrono::steady_clock::now();
  const size_t input_size = line.size();
  // Input string for the device
  char *device_input = nullptr;
  char *nonces = nullptr;
  char *result_nonce = nullptr;

  // Output string by the device read by host
  char *g_out = nullptr;
  unsigned char *g_hash_out = nullptr;
  int *g_found = nullptr;

  char *result_nonce_host = (char *)malloc(14 * sizeof(char));

  char g_out_host[input_size + 32 + 1];
  unsigned char *g_hash_out_host[13 + 1];
  int g_found_host = 0;

  pre_sha256();

  // Create the input string for the device
  hipMalloc(&device_input, input_size + 1);

  hipMemcpy(device_input, line.c_str(), input_size + 1,
             hipMemcpyHostToDevice);
  hipMalloc(&result_nonce, 10 + 1);
  hipMalloc(&g_out, input_size + 32 + 1);
  hipMalloc(&g_hash_out, 32);
  hipMalloc(&g_found, sizeof(int));

  hipMemcpy(g_found, &g_found_host, sizeof(int), hipMemcpyHostToDevice);
  hipError_t err_result = hipGetLastError();
  if (err_result != hipSuccess) {
    throw std::runtime_error("Device error \n" +
                             std::string(hipGetErrorString(err_result)));
  }

  while (!g_found_host) {

    fflush(stdout);
    sha256_kernel<<<dim3(1, NUMBLOCKS), BLOCK_SIZE>>>(
        g_out, g_hash_out, g_found, result_nonce, device_input, input_size,
        difficulty, offset);
    hipMemcpy(&g_found_host, g_found, sizeof(int), hipMemcpyDeviceToHost);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      throw std::runtime_error("Device error");
    }
    const hipError_t err_result = hipGetLastError();
    if (err_result != hipSuccess) {
      throw std::runtime_error("Device error \n" +
                               std::string(hipGetErrorString(err_result)));
    }
    nonce += NUMBLOCKS * BLOCK_SIZE;
  }
  auto end_time = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(
                      end_time - start_time)
                      .count();
  // printf("speed : %f hash/s\n", nonce / (duration / 1000.0));
  // printf("duration: %d ms\n", duration);
  // printf(line.c_str(), result_nonce, difficulty);
  hipMemcpy(result_nonce_host, result_nonce, 10 + 1, hipMemcpyDeviceToHost);
  err_result = hipGetLastError();
  if (err_result != hipSuccess) {
    throw std::runtime_error("Device error  Cioy errir \n" +
                             std::string(hipGetErrorString(err_result)));
  }

  // std::string result(result_nonce_host);
  // printf("\nresult nonce: %s\n", result.c_str());

  hipFree(g_out);
  hipFree(g_hash_out);
  hipFree(g_found);
  hipFree(device_input);

  // hipDeviceReset();
  // for (int i = 0; i <= line.size(); ++i) {
  //   if (i >= offset && i < offset + 10) {
  //     line[i] = result[i - offset];
  //   }
  // }
  // printf("input is %s \n", line.c_str());
  return result_nonce_host;
}
// int main() {

//   t_last_updated = std::chrono::high_resolution_clock::now();
//   std::ifstream file("test.txt");
//   std::string line;
//   if (file.is_open()) {
//     getline(file, line);
//     file.close();
//   } else {
//     std::cout << "无法打开文件" << std::endl;
//     exit(1);
//   }
//   solve(line, 4);
//   return 0;
// }

extern "C" {
const char *solve_noss(char *input, int difficulty) {
  std::string input_value(input);
  return solve(input_value, difficulty);
}
}